#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<stdbool.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Result from last compute of world.
extern unsigned char *g_resultData;

// Current state of world. 
extern unsigned char *g_data;

// ----- SAVE RECEIVING ROWS FROM OTHER GPUS ----- //
// "Above" row
extern unsigned char *g_aboveRow;

// "Below" row 
extern unsigned char *g_belowRow;

// "Above" row
extern unsigned char *g_resultAboveRow;

// "Below" row 
extern unsigned char *g_resultBelowRow;

// ----- DECLARE KERNEL ----- //
__global__ void HL_kernel(unsigned int worldWidth, unsigned int worldHeight);


// Define number of Processors
int cudaDeviceCount;
hipError_t cE; 


__global__ void HL_kernel( unsigned char* d_data, unsigned char* d_resultData, unsigned char* d_aboveRow, unsigned char* d_belowRow, unsigned int worldWidth, unsigned int worldHeight){
	size_t index;

    // Loop over the threads
    for(index = blockIdx.x * blockDim.x + threadIdx.x; index < worldWidth*worldHeight; index += blockDim.x * gridDim.x){

        // Allocate space
        int y0 = ((index + worldHeight - 1) % worldHeight) * worldWidth;
        int y1 = index * worldWidth;
        int y2 = ((index + 1) % worldHeight) * worldWidth;

        // Get the current block and thread
        int x;

        // Loop over corresponding COLUMNS
	        for (x = 0; x < worldWidth; ++x){

            // Set current column, left column, and right column
            int x1 = x;
            int x0 = (x1 + worldWidth - 1) % worldWidth; 
            int x2 = (x1 + 1) % worldWidth;

            // Get the status of the current cell to determine logic of life span
            int is_alive = d_data[x1+y1];

            // Count the number of alive neighbors
            int num_alive = 0;

            // Check above and below row cases
            if (x1+y1 < worldWidth) {
                num_alive = d_aboveRow[x0] + d_aboveRow[x1] + d_aboveRow[x2] + d_data[x0+y1] + d_data[x2+y1] + d_data[x0+y2] + d_data[x1+y2] + d_data[x2+y2];
            }
            else if (x1+y1 > worldWidth*worldHeight - worldWidth - 1) {
                num_alive = d_data[x0+y0] + d_data[x1+y0] + d_data[x2+y0] + d_data[x0+y1] + d_data[x2+y1] + d_belowRow[x0] + d_belowRow[x1] + d_belowRow[x2];
            }
            else {
                num_alive = d_data[x0+y0] + d_data[x1+y0] + d_data[x2+y0] + d_data[x0+y1] + d_data[x2+y1] + d_data[x0+y2] + d_data[x1+y2] + d_data[x2+y2];
            }

            // Logic for updating values
            if (is_alive == 1){
                // Cell is alive!
                if (num_alive < 2){
                    // Underpopulated
                    d_resultData[x1+y1] = 0;
                }
                else if (num_alive == 2 || num_alive == 3){
                    // Just the right amount of neighbors
                    d_resultData[x1+y1] = 1;
                }
                else {
                    // Overpopulated
                    d_resultData[x1+y1] = 0;
                }
            }
            else {
                // Cell is dead :(
                if (num_alive == 3 || num_alive == 6) {
                    // #Resurrected
                    d_resultData[x1+y1] = 1;
                }
                else {
                    // We stay dead
                    d_resultData[x1+y1] = 0;
                }
            }
	    } 
    } 

    // ----- SWAP DATA IN ABOVE ROWS AND BELOW ROWS ----- //
    int j;
    for(j = 0; j < worldWidth; j++){
        d_aboveRow[j] = d_resultData[j];
        d_belowRow[j] = d_resultData[j + worldWidth*(worldHeight - 1)];
    }

    // Synchronize the threads?
    __syncthreads();

}

extern "C" void HL_initMaster( unsigned int pattern, size_t worldWidth, size_t worldHeight, int myrank, int cudaDeviceCount )
{
    
    // INITIALIZE THE CUDA WORLD
    if( (cE = hipGetDeviceCount( &cudaDeviceCount)) != hipSuccess )
    {
    printf(" Unable to determine cuda device count, error is %d, count is %d\n", cE, cudaDeviceCount );
    exit(-1);
    }
    if( (cE = hipSetDevice( myrank % cudaDeviceCount )) != hipSuccess )
    {
    printf(" Unable to have myrank %d set to cuda device %d, error is %d \n", myrank, (myrank % cudaDeviceCount), cE);
    exit(-1); 
    }

    switch(pattern)
    {
    case 0:
	HL_initAllZeros( worldWidth, worldHeight, myrank, cudaDeviceCount );
	break;
	
    case 1:
	HL_initAllOnes( worldWidth, worldHeight, myrank, cudaDeviceCount  );
	break;
	
    case 2:
	HL_initOnesInMiddle( worldWidth, worldHeight, myrank, cudaDeviceCount  );
	break;
	
    case 3:
	HL_initOnesAtCorners( worldWidth, worldHeight, myrank, cudaDeviceCount  );
	break;

    case 4:
	HL_initSpinnerAtCorner( worldWidth, worldHeight, myrank, cudaDeviceCount  );
	break;

    case 5:
	HL_initReplicator( worldWidth, worldHeight, myrank, cudaDeviceCount  );
	break;
	
    default:
	printf("Pattern %u has not been implemented \n", pattern);
	exit(-1);
    }
}


extern "C" void HL_kernelLaunch( unsigned char** d_data, unsigned char** d_resultData, unsigned char** d_aboveRow, unsigned char** d_belowRow, int block_count, int thread_count, unsigned int worldWidth, unsigned int worldHeight, int myrank){
    
    // Call the kernel
    HL_kernel<<<block_count,thread_count>>>(*d_data, *d_resultData, *d_aboveRow, *d_belowRow, worldWidth, worldHeight);

    hipDeviceSynchronize();
}


extern "C" void freeCudaArrays(int myrank){
    hipFree(g_data);
    hipFree(g_resultData);
    hipFree(g_aboveRow);
    hipFree(g_belowRow);
}