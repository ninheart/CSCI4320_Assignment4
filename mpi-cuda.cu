#include<stdio.h>
#include<stdlib.h>
#include<unistd.h>
#include<stdbool.h>
#include <hip/hip_runtime.h>


// Result from last compute of world.
extern unsigned char *g_resultData;

// Current state of world. 
extern unsigned char *g_data;

// "Above" row
extern unsigned char *g_above_row;

// "Below" row 
extern unsigned char *g_below_row;

static inline void HL_initAllZeros( size_t worldWidth, size_t worldHeight )
{
    size_t g_dataLength = worldWidth * worldHeight;

    // calloc init's to all zeros
    hipMallocManaged( &g_data, (g_dataLength * sizeof(unsigned char)));
    hipMallocManaged( &g_resultData, (g_dataLength * sizeof(unsigned char))); 

    size_t i = 0;
    for (i = 0; i < g_dataLength; i++)
    {
        g_data[i] = 0;
        g_resultData[i] = 0;
    }
}

static inline void HL_initReplicator( size_t worldWidth, size_t worldHeight )
{
    HL_initAllZeros(worldWidth, worldHeight);

    size_t x, y;

    x = worldWidth/2;
    y = worldHeight/2;
    
    g_data[x + y*worldWidth + 1] = 1; 
    g_data[x + y*worldWidth + 2] = 1;
    g_data[x + y*worldWidth + 3] = 1;
    g_data[x + (y+1)*worldWidth] = 1;
    g_data[x + (y+2)*worldWidth] = 1;
    g_data[x + (y+3)*worldWidth] = 1; 
}

__global__ void HL_kernel(const unsigned char* d_data, unsigned char* d_resultData,
                            unsigned int worldWidth, unsigned int worldHeight)
{
    size_t index = blockIdx.x *blockDim.x + threadIdx.x;

    for (; index < worldWidth*worldHeight; index += blockDim.x * gridDim.x)
    {
        // get the x and y coords from the index in the flattened world
        size_t y = (size_t) index / worldWidth;

        size_t y0 = ((y + worldHeight - 1) % worldHeight) * worldWidth;
        size_t y1 = y * worldWidth;
        size_t y2 = ((y + 1) % worldHeight) * worldWidth;

        size_t x1 = index % worldWidth;

        size_t x0 = (x1 + worldWidth - 1) % worldWidth;
        size_t x2 = (x1 + 1) % worldWidth;

        // The rest is similar to the serial code, with the adjacent cells checked
        unsigned int aliveCells = d_data[x0 + y0] + d_data[x1 + y0] + d_data[x2 + y0]
                                    + d_data[x0 + y1] + d_data[x2 + y1]
                                    + d_data[x0 + y2] + d_data[x1 + y2] + d_data[x2 + y2];;

        d_resultData[x1 + y1] = (aliveCells == 3) || (aliveCells == 6 && !d_data[x1 + y1])
          || (aliveCells == 2 && d_data[x1 + y1]) ? 1 : 0;
    }
}

extern "C" void HL_initMaster( unsigned int pattern, size_t worldWidth, size_t worldHeight, int myrank)
{
    // Set device
    int cudaDeviceCount;
    hipError_t cE; 
    if( (cE = hipGetDeviceCount( &cudaDeviceCount)) != hipSuccess )
    {
        printf(" Unable to determine cuda device count, error is %d, count is %d\n", cE, cudaDeviceCount );
        exit(-1);
    }
    if( (cE = hipSetDevice( myrank % cudaDeviceCount )) != hipSuccess )
    {
        printf(" Unable to have myrank %d set to cuda device %d, error is %d \n", myrank, (myrank % cudaDeviceCount), cE);
        exit(-1); 
    }

    // INITIALIZE THE CUDA WORLD
    HL_initReplicator( worldWidth, worldHeight );
}


extern "C" void HL_kernelLaunch( unsigned char** d_data, unsigned char** d_resultData, 
        unsigned char * next_above_row, unsigned char * next_below_row, 
        int block_count, int thread_count, 
        unsigned int worldWidth, unsigned int worldHeight, 
        int myrank){
    
    // load back into device
    hipMemcpy(g_data, next_above_row, worldWidth, hipMemcpyHostToDevice);
    hipMemcpy(g_data+(worldHeight- 1) * worldWidth, next_below_row, worldWidth, hipMemcpyHostToDevice);

    // Call the kernel
    HL_kernel<<<block_count,thread_count>>>(*d_data, *d_resultData, worldWidth, worldHeight);
    hipDeviceSynchronize();

    //load from device to host
    hipMemcpy(g_above_row, g_data, worldWidth, hipMemcpyDeviceToHost);
    hipMemcpy(g_below_row, g_data+(worldHeight - 1) * worldWidth, worldWidth, hipMemcpyDeviceToHost);
}


extern "C" void freeCudaArrays(int myrank){
    hipFree(g_data);
    hipFree(g_resultData);
}